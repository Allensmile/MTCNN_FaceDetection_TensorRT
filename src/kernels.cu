#include "hip/hip_runtime.h"
#include "kernels.h"
#include <thrust/device_vector.h>
using namespace cv::cuda;
__global__ void image2Matrix_kernel(int width, int height,  PtrStepSz<uchar3> image, float* matrix){

    const int w = blockIdx.x;
    const int h = blockIdx.y;
    
    if (w < width && h < height)
    {
        uchar3 v = image(h,w);
        *(matrix + 0*height*width + h*width + w) = (float(v.z)-127.5)*0.0078125;
        *(matrix + 1*height*width + h*width + w) = (float(v.y)-127.5)*0.0078125;
        *(matrix + 2*height*width + h*width + w) = (float(v.x)-127.5)*0.0078125;
    }

}
void gpu_image2Matrix(int width, int height,  GpuMat & image, float* matrix)
{
    /*
        image : input image in GpuMat format, WHC arrangement and BGR order
        matrix: gpu float array, CHW and RGB order
    */
    dim3 block(width, height); // width * height blocks, 1 thread each
    image2Matrix_kernel<<<block,1>>>(width,height,image,matrix);
}



//__global__ void generatebox_kernel(int width, int height, float * scores, float * location, float pthreshold )
//{
//    const int w = blockIdx.x;
//    const int h = blockIdx.y;
//    if(w<width && h< height)
//    {
//        float score  = *(scores + width*height + w*height+ h );
//        if(score > pthreshold)
//        {
//
//        }
//    }
//
//
//}
//void gpu_generatebox(int width , int height , void * score, void * location, float scale, float pthreshold)
//{
//
//    int stride = 2;
//    int cellsize = 12;
//    int count = 0;
//    //score p
//    void *p = (float*)score + width * height;
//    void *plocal = (float*)location;
//    struct Bbox bbox;
//    struct orderScore order;
//    for (int row = 0; row < score->height; row++) {
//        for (int col = 0; col < score->width; col++) {
//            if (*p > Pthreshold) {
//                bbox.score = *p;
//                order.score = *p;
//                order.oriOrder = count;
//                bbox.x1 = round((stride * row + 1) / scale);
//                bbox.y1 = round((stride * col + 1) / scale);
//                bbox.x2 = round((stride * row + 1 + cellsize) / scale);
//                bbox.y2 = round((stride * col + 1 + cellsize) / scale);
//                bbox.exist = true;
//                bbox.area = (bbox.x2 - bbox.x1) * (bbox.y2 - bbox.y1);
//                for (int channel = 0; channel < 4; channel++)
//                    bbox.regreCoord[channel] = *(plocal + channel * location->width * location->height);
//                boundingBox_.push_back(bbox);
//                bboxScore_.push_back(order);
//                count++;
//            }
//            p++;
//            plocal++;
//        }
//    }
//    dim3 block(width, height);
//    generatebox_kernel(width, height, score, location, pthreshold);

//}
